#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BlockSize 32

const int INF = ((1 << 30) - 1);
/* start some declaration */

int* Dist; 
int n , m , new_n;
/* end the declaration */

int ceil(int a, int b) { return (a + b - 1) / b; }

__global__ void phase1(int* Dist_d, int round_cnt, int new_n){
    __shared__ int store[BlockSize][BlockSize];
    int x_real = threadIdx.x + round_cnt * BlockSize;
    int y_real = threadIdx.y + round_cnt * BlockSize;
    int x = threadIdx.x;
    int y = threadIdx.y;

    /*if(x_real >= new_n || y_real  >= new_n){
        return;
    }*/
    store[x][y] = Dist_d[x_real * new_n + y_real];
    
    __syncthreads();
    for(int i = 0 ; i < BlockSize ; i++){
        store[x][y] = min(store[x][i] + store[i][y] , store[x][y]);
        __syncthreads();
    }
    Dist_d[x_real * new_n + y_real] = store[x][y];  
}

__global__ void phase2(int* Dist_d, int round_cnt, int new_n){
    __shared__ int store[BlockSize][BlockSize];
    __shared__ int vertical[BlockSize][BlockSize];
    __shared__ int herizonal[BlockSize][BlockSize];
    int x = threadIdx.x;
    int y = threadIdx.y;
    int block_num = blockIdx.y;
    int x_ver = threadIdx.x + round_cnt * BlockSize;
    int x_her = threadIdx.x + block_num * BlockSize;
    int y_ver = threadIdx.y + block_num * BlockSize;
    int y_her = threadIdx.y + round_cnt * BlockSize;
    // if duplicate with the phase 1
    if(block_num == round_cnt ){
        return;
    }
    // since we calculate both vertical and herizonal at once ,we can't delete both given one doesn't fulfill
    store[x][y] = Dist_d[x_ver * new_n + y_her];
    herizonal[x][y] = Dist_d[ x_her * new_n + y_her];
    vertical[x][y] = Dist_d[ x_ver * new_n + y_ver ];
    __syncthreads();
    #pragma unroll 32
    for(int i = 0 ; i < BlockSize ; i++){
        vertical[x][y] = min( store[x][i] + vertical[i][y] , vertical[x][y]);
        herizonal[x][y] =  min( herizonal[x][i] + store[i][y] , herizonal[x][y]);
    }
    Dist_d[x_her * new_n + y_her] = herizonal[x][y];
    Dist_d[x_ver * new_n + y_ver] = vertical[x][y];
}

__global__ void phase3(int* Dist_d, int round_cnt, int new_n){
    __shared__ int store[BlockSize][BlockSize];
    __shared__ int vertical[BlockSize][BlockSize];
    __shared__ int herizonal[BlockSize][BlockSize];
    int block_x = blockIdx.x;
    int block_y = blockIdx.y;
    int x = threadIdx.x;
    int y = threadIdx.y;
    if(block_x == round_cnt || block_y == round_cnt){
        return;
    }
    int x_real = block_x * BlockSize + x;
    int y_real = block_y * BlockSize + y;
    int x_her = x_real;
    int y_her = round_cnt * BlockSize + y;
    int x_ver = round_cnt * BlockSize + x;
    int y_ver = y_real;
    if(x_real >= new_n || y_real >= new_n){
        return;
    }
    store[x][y] = Dist_d[x_real * new_n + y_real];
    vertical[x][y] = Dist_d[x_ver * new_n + y_ver];
    herizonal[x][y] = Dist_d[x_her * new_n + y_her];
    __syncthreads();
    #pragma unroll 32
    for(int i = 0 ; i < BlockSize ; i++){
        store[x][y] = min(herizonal[x][i] + vertical[i][y] , store[x][y]);
    }
    Dist_d[x_real * new_n + y_real] = store[x][y] ;
}

void input(char* infile) {

    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    new_n = ceil(n , BlockSize) * BlockSize; // for the purpose to make coalsememory
    Dist = (int*) malloc(new_n * new_n *sizeof(int));

    for (int i = 0; i < new_n; ++i) {
        for (int j = 0; j < new_n ; ++j) {
            if (i == j) {
                Dist[i * new_n + j] = 0;
            } else {
                Dist[i * new_n + j] = INF;
            }
        }
    }


    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0] * new_n + pair[1]] = pair[2];
    }

    /*for(int i = 0 ; i < n ; i++){
        for(int j = 0 ; j < n ; j++){
            printf("%d " , Dist[i * new_n + j]);
        }
        printf("\n");
    }*/
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; ++i) {
		fwrite(Dist + i * new_n , sizeof(int), n , outfile);
	}
    for(int i = 0 ; i < n ; i++){
        for(int j = 0 ; j < n ; j++){
            printf("%d " , Dist[i * new_n + j]);
        }
        printf("\n");
    }
    fclose(outfile);
}



int main(int argc, char* argv[]) {
    input(argv[1]);
    // int BlockSize = 512;
    int* deviceDist;
    hipMalloc(&deviceDist, new_n * new_n * sizeof(int));
	hipMemcpy(deviceDist, Dist, new_n * new_n * sizeof(int), hipMemcpyHostToDevice);

    int total_block_num = new_n / BlockSize;
    dim3 block_num1(1, 1);
    dim3 block_num2(1, total_block_num);
    dim3 block_num3(total_block_num, total_block_num);
    dim3 thread_each_block(BlockSize,BlockSize);
    // printf("the total block : %d \n" , total_block_num);
    // printf("the n is %d and the new_n is %d \n" ,  n , new_n);
    
    for(int i = 0 ; i < total_block_num ; i++){
        phase1<<<block_num1 , thread_each_block>>>(deviceDist , i , new_n); // phase 1 
        phase2<<<block_num2 , thread_each_block>>>(deviceDist , i , new_n); // phase 2
        phase3<<<block_num3 , thread_each_block>>>(deviceDist , i , new_n); // phase 3
    }
    hipMemcpy(Dist , deviceDist, new_n * new_n * sizeof(int), hipMemcpyDeviceToHost); 
    output(argv[2]);
    return 0;
}