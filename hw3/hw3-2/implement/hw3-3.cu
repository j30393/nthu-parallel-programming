#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BlockSize 64
#define Half 32
const int INF = ((1 << 30) - 1);
/* start some declaration */

int* Dist; 
int n , m , new_n;
/* end the declaration */

int ceil(int a, int b) { return (a + b - 1) / b; }

__device__ void autogenerate(int* Dist_d , int* target , int x , int y , int x_d  , int y_d , int new_n){

    target[y * BlockSize + x  ] = Dist_d[y_d * new_n + x_d];
    target[y * BlockSize + x + 1 ] = Dist_d[y_d * new_n + x_d + 1];
    target[y * BlockSize + x + 32] = Dist_d[y_d * new_n + x_d + 32];
    target[y * BlockSize + x + 33 ] = Dist_d[y_d * new_n + x_d + 33];
}

__device__ void write_back(int* Dist_d , int* target , int x , int y , int x_d  , int y_d , int new_n){
    Dist_d[y_d * new_n + x_d] = target[y * BlockSize + x ];
    Dist_d[y_d * new_n + x_d + 1] = target[y * BlockSize + x + 1];
    Dist_d[y_d * new_n + x_d + 32] = target[y * BlockSize + x + 32];
    Dist_d[y_d * new_n + x_d + 33] = target[y * BlockSize + x + 33];
}

__global__ void phase1(int* Dist_d, int round_cnt, int new_n){
    // input shape (16,64)
    __shared__ int store[BlockSize * BlockSize];
    int x = threadIdx.x * 2;
    int y = threadIdx.y;
    int x_real = x + round_cnt * BlockSize;
    int y_real = threadIdx.y + round_cnt * BlockSize;
    autogenerate(Dist_d ,store ,x ,y, x_real , y_real , new_n);

    __syncthreads();
    #pragma unroll 32
    for(int i = 0 ; i < BlockSize ; i++){
        store[y * BlockSize + x] = min(store[y * BlockSize + i] + store[i* BlockSize + x] , store[y * BlockSize + x]);
        store[y * BlockSize + x + 1] = min(store[y * BlockSize + i ] + store[i* BlockSize + x + 1] , store[y * BlockSize + x + 1]);
        store[y * BlockSize + x + 32] = min(store[y * BlockSize + i ] + store[i* BlockSize + x + 32] , store[y * BlockSize + x + 32]);
        store[y * BlockSize + x + 33] = min(store[y * BlockSize + i ] + store[i* BlockSize + x + 33] , store[y * BlockSize + x + 33]);
        __syncthreads();
    }
    write_back(Dist_d , store ,x ,y, x_real , y_real , new_n);
}

__global__ void phase2(int* Dist_d, int round_cnt, int new_n){
    __shared__ int store[BlockSize * BlockSize];
    __shared__ int vertical[BlockSize * BlockSize];
    __shared__ int herizonal[BlockSize * BlockSize];
    int x = threadIdx.x * 2;
    int y = threadIdx.y;
    int block_num = blockIdx.y;
    
    int x_ver = x + round_cnt * BlockSize;
    int x_her = x + block_num * BlockSize;
    int y_ver = y + block_num * BlockSize;
    int y_her = y + round_cnt * BlockSize;
    
    // if duplicate with the phase 1
    /*if(block_num == round_cnt ){
        return;
    }*/
    // since we calculate both vertical and herizonal at once ,we can't delete both given one doesn't fulfill
    autogenerate(Dist_d ,store , x , y , x_ver , y_her , new_n);
    autogenerate(Dist_d ,herizonal , x , y , x_her , y_her , new_n);
    autogenerate(Dist_d ,vertical , x , y , x_ver , y_ver , new_n);

    __syncthreads();
    #pragma unroll 32
    for(int i = 0 ; i < BlockSize ; i++){
        vertical[y * BlockSize + x] = min( vertical[y * BlockSize + i] + store[i * BlockSize + x] , vertical[y * BlockSize + x]);
        vertical[y * BlockSize + x + 1] = min( vertical[y * BlockSize + i] + store[i * BlockSize + x + 1] , vertical[y * BlockSize + x + 1]);
        vertical[y * BlockSize + x + 32] = min( vertical[y * BlockSize + i] + store[i * BlockSize + x + 32] , vertical[y * BlockSize + x + 32]);
        vertical[y * BlockSize + x + 33] = min( vertical[y * BlockSize + i] + store[i * BlockSize + x + 33] , vertical[y * BlockSize + x + 33]);

        herizonal[y * BlockSize + x] =  min( store[y * BlockSize + i] + herizonal[i * BlockSize + x] , herizonal[y * BlockSize + x]);
        herizonal[y * BlockSize + x + 1] =  min( store[y * BlockSize + i] + herizonal[i * BlockSize + x + 1] , herizonal[y * BlockSize + x + 1]);
        herizonal[y * BlockSize + x + 32] =  min( store[y * BlockSize + i] + herizonal[i * BlockSize + x + 32] , herizonal[y * BlockSize + x + 32]);
        herizonal[y * BlockSize + x + 33] =  min( store[y * BlockSize + i] + herizonal[i * BlockSize + x + 33] , herizonal[y * BlockSize + x + 33]);
    }

    write_back(Dist_d ,herizonal , x , y , x_her , y_her , new_n);
    write_back(Dist_d ,vertical , x , y , x_ver , y_ver , new_n);
}

__global__ void phase3(int* Dist_d, int round_cnt, int new_n){
    __shared__ int store[BlockSize * BlockSize];
    __shared__ int vertical[BlockSize * BlockSize];
    __shared__ int herizonal[BlockSize * BlockSize];
    int x = threadIdx.x * 2;
    int y = threadIdx.y;
    /*if(block_x == round_cnt || block_y == round_cnt){
        return;
    }*/
    int x_her = blockIdx.x * BlockSize + x;
    int y_her = round_cnt * BlockSize + y;
    int x_ver = round_cnt * BlockSize + x;
    int y_ver =  blockIdx.y * BlockSize + y;
    /*if(x_real >= new_n || y_real >= new_n){
        return;
    }*/
    autogenerate(Dist_d , store , x , y , x_her , y_ver , new_n);
    autogenerate(Dist_d , herizonal ,x , y, x_her , y_her , new_n);
    autogenerate(Dist_d , vertical , x , y ,x_ver , y_ver , new_n);

    __syncthreads();
    #pragma unroll 32
    for(int i = 0 ; i < BlockSize ; i++){
        store[y * BlockSize + x] = min(vertical[y * BlockSize + i] + herizonal[i * BlockSize + x] , store[y * BlockSize + x]);
        store[y * BlockSize + x + 1] = min(vertical[y * BlockSize + i] + herizonal[i * BlockSize + x + 1] , store[y * BlockSize + x + 1]);
        store[y * BlockSize + x + 32] = min(vertical[y * BlockSize + i] + herizonal[i * BlockSize + x + 32] , store[y * BlockSize + x + 32]);
        store[y * BlockSize + x + 33] = min(vertical[y * BlockSize + i] + herizonal[i * BlockSize + x + 33] , store[y * BlockSize + x + 33]);
    }
    
    write_back(Dist_d , store , x , y , x_her , y_ver , new_n);
}


void input(char* infile) {

    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    new_n = ceil(n , BlockSize) * BlockSize; // for the purpose to make coalsememory
    Dist = (int*) malloc(new_n * new_n *sizeof(int));

    for (int i = 0; i < new_n; ++i) {
        for (int j = 0; j < new_n ; ++j) {
            Dist[i * new_n + j] = (i == j) ? 0 :INF;
        }
    }


    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0] * new_n + pair[1]] = pair[2];
    }

    /*for(int i = 0 ; i < n ; i++){
        for(int j = 0 ; j < n ; j++){
            printf("%d " , Dist[i * new_n + j]);
        }
        printf("\n");
    }*/
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; ++i) {
		fwrite(Dist + i * new_n , sizeof(int), n , outfile);
	}
    /*for(int i = 0 ; i < n ; i++){
        for(int j = 0 ; j < n ; j++){
            printf("%d " , Dist[i * new_n + j]);
        }
        printf("\n");
    }*/
    fclose(outfile);
}



int main(int argc, char* argv[]) {
    input(argv[1]);
    // int BlockSize = 512;
    int* deviceDist;
    printf("%d \n", new_n);
    size_t size_ = sizeof(int) *new_n * new_n;
    hipHostRegister(Dist,size_, hipHostRegisterDefault);
    hipMalloc(&deviceDist, size_);
	hipMemcpy(deviceDist, Dist, size_, hipMemcpyHostToDevice);

    int total_block_num = new_n / BlockSize;
    dim3 block_num1(1, 1);
    dim3 block_num2(1, total_block_num);
    dim3 block_num3(total_block_num, total_block_num);
    // dim3 thread_each_block(BlockSize,BlockSize);
    dim3 thread_each_block(16 , BlockSize);
    //dim3 thread_32(32,32);
    // printf("the total block : %d \n" , total_block_num);
    // printf("the n is %d and the new_n is %d \n" ,  n , new_n);
    
    for(int i = 0 ; i < total_block_num ; i++){
        phase1<<<block_num1 , thread_each_block>>>(deviceDist , i , new_n); // phase 1 
        phase2<<<block_num2 , thread_each_block>>>(deviceDist , i , new_n); // phase 2
        phase3<<<block_num3 , thread_each_block>>>(deviceDist , i , new_n); // phase 3
    }
    hipMemcpy(Dist , deviceDist, size_, hipMemcpyDeviceToHost); 
    output(argv[2]);
    return 0;
}