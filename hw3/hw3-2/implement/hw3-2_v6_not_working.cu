#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BlockSize 64
#define Half 32
const int INF = ((1 << 30) - 1);
/* start some declaration */

int* Dist; 
int n , m , new_n;
/* end the declaration */

int ceil(int a, int b) { return (a + b - 1) / b; }

__device__ void autogenerate(int* Dist_d , int* target , int x , int y , int new_n){
    target[y * BlockSize + x ] = Dist_d[y * new_n + x];
    target[y * BlockSize + (x + Half) ] = Dist_d[y * new_n + (x + Half)];
    target[(y + Half) * BlockSize + x ] = Dist_d[(y + Half) * new_n + x ];
    target[(y + Half) * BlockSize + (x + Half) ] = Dist_d[(y + Half) * new_n + (x + Half)];
}

__global__ void phase1(int* Dist_d, int round_cnt, int new_n){
    // input shape (32,32)
    __shared__ int store[BlockSize * BlockSize];
    int x = threadIdx.x;
    int y = threadIdx.y;
    int x_real = threadIdx.x + round_cnt * BlockSize;
    int y_real = threadIdx.y + round_cnt * BlockSize;
    autogenerate(Dist_d ,store , x_real , y_real , new_n);

    __syncthreads();
    #pragma unroll 32
    for(int i = 0 ; i < BlockSize ; i++){
        store[y * BlockSize + x] = min(store[y * BlockSize + i] + store[i* BlockSize + x] , store[y * BlockSize + x]);
        store[(y + Half) * BlockSize + x] = min(store[(y + Half) * BlockSize + i] + store[i* BlockSize + x] , store[(y + Half) * BlockSize  + x]);
        store[ y  * BlockSize + (x+ Half)] = min(store[y * BlockSize + i  ] + store[ i * BlockSize + (x+ Half)] , store[ y  * BlockSize + (x+ Half)]);
        store[(y + Half)  * BlockSize + (x+ Half)] = min(store[(y + Half) * BlockSize + i  ] + store[ i * BlockSize + (x+ Half)] , store[ (y + Half) * BlockSize + (x+ Half)]);
        __syncthreads();
    }
    Dist_d[y_real * new_n + x_real] = store[y * BlockSize + x];
    Dist_d[ ( y_real + Half )* new_n + x_real] = store[(y + Half) * BlockSize + x];
    Dist_d[y_real * new_n + (x_real + Half) ] = store[y * BlockSize +(x + Half)];
    Dist_d[( y_real + Half )* new_n + x_real + Half ] = store[(y + Half) * BlockSize + x + Half];
}

__global__ void phase2(int* Dist_d, int round_cnt, int new_n){
    __shared__ int store[BlockSize * BlockSize];
    __shared__ int vertical[BlockSize * BlockSize];
    __shared__ int herizonal[BlockSize * BlockSize];
    int x = threadIdx.x;
    int y = threadIdx.y;
    int block_num = blockIdx.y;
    
    int x_ver = threadIdx.x + round_cnt * BlockSize;
    int x_her = threadIdx.x + block_num * BlockSize;
    int y_ver = y + block_num * BlockSize;
    int y_her = y + round_cnt * BlockSize;
    
    // if duplicate with the phase 1
    /*if(block_num == round_cnt ){
        return;
    }*/
    // since we calculate both vertical and herizonal at once ,we can't delete both given one doesn't fulfill
    autogenerate(Dist_d ,store , x_ver , y_her , new_n);
    autogenerate(Dist_d ,herizonal , x_her , y_her , new_n);
    autogenerate(Dist_d ,vertical , x_ver , y_ver , new_n);

    __syncthreads();
    #pragma unroll 32
    for(int i = 0 ; i < BlockSize ; i++){
        vertical[y * BlockSize + x] = min( store[y * BlockSize + i] + vertical[i * BlockSize + x] , vertical[y * BlockSize + x]);
        vertical[(y + Half) * BlockSize + x] = min( store[(y + Half) * BlockSize + i] + vertical[i * BlockSize + x] , vertical[(y + Half) * BlockSize + x]);
        vertical[y * BlockSize + (x + Half)] = min( store[y * BlockSize + i] + vertical[i * BlockSize + (x + Half)] , vertical[y * BlockSize + (x + Half)]);
        vertical[(y + Half) * BlockSize +(x + Half)] = min( store[(y + Half) * BlockSize + i] + vertical[i * BlockSize + (x + Half)] , vertical[(y + Half) * BlockSize + (x + Half)]);

        herizonal[y * BlockSize + x] =  min( herizonal[y * BlockSize + i] + store[i * BlockSize + x] , herizonal[y * BlockSize + x]);
        herizonal[(y + Half) * BlockSize + x] =  min( herizonal[(y + Half) * BlockSize + i] + store[i * BlockSize + x] , herizonal[(y + Half) * BlockSize + x]);
        herizonal[y * BlockSize + (x + Half)] =  min( herizonal[y * BlockSize + i] + store[i * BlockSize + (x + Half)] , herizonal[y * BlockSize + (x + Half)]);
        herizonal[(y + Half) * BlockSize + (x + Half)] =  min( herizonal[(y + Half) * BlockSize + i] + store[i * BlockSize + (x + Half)] , herizonal[(y + Half) * BlockSize + (x + Half)]);
    }

    Dist_d[y_her * new_n + x_ver] = store[y * BlockSize + x] ;
    Dist_d[(y_her + Half) * new_n + x_ver] = store[(y + Half)* BlockSize + x] ;
    Dist_d[y_her * new_n + x_ver + Half] = store[y * BlockSize + ( x + Half)] ;
    Dist_d[(y_her + Half) * new_n + ( x_ver + Half )] = store[(y + Half) * BlockSize + (x + Half) ] ;

    Dist_d[ y_her * new_n + x_ver ] = herizonal[y * BlockSize + x ] ;
    Dist_d[ (y_her + Half) * new_n + x_ver ] = herizonal[(y + Half)* BlockSize + x ];
    Dist_d[ y_her * new_n + (x_ver + Half) ] = herizonal[y * BlockSize + ( x + Half)] ;
    Dist_d[ ( y_her + Half) * new_n + ( x_ver + Half) ] = herizonal[(y + Half) * BlockSize + (x + Half) ];

    Dist_d[ y_ver * new_n + x_ver ] = vertical[y * BlockSize + x];
    Dist_d[ (y_ver+ Half) * new_n + x_ver ] = vertical[(y + Half)* BlockSize + x ];
    Dist_d[ y_ver * new_n + ( x_ver+ Half ) ] = vertical[y * BlockSize + ( x + Half)];
    Dist_d[ ( y_ver + Half) * new_n +  ( x_ver + Half) ] = vertical[(y + Half) * BlockSize + (x + Half) ];
}

__global__ void phase3(int* Dist_d, int round_cnt, int new_n){
    __shared__ int store[BlockSize * BlockSize];
    __shared__ int vertical[BlockSize * BlockSize];
    __shared__ int herizonal[BlockSize * BlockSize];
    int block_x = blockIdx.x;
    int block_y = blockIdx.y;
    int x = threadIdx.x;
    int y = threadIdx.y;
    /*if(block_x == round_cnt || block_y == round_cnt){
        return;
    }*/
    int x_real = block_x * BlockSize + x;
    int y_real = block_y * BlockSize + y;
    int x_her = x_real;
    int y_her = round_cnt * BlockSize + y;
    int x_ver = round_cnt * BlockSize + x;
    int y_ver = y_real;
    /*if(x_real >= new_n || y_real >= new_n){
        return;
    }*/
    autogenerate(Dist_d , store , x_real , y_real , new_n);
    autogenerate(Dist_d , herizonal , x_her , y_her , new_n);
    autogenerate(Dist_d , vertical , x_ver , y_ver , new_n);

    __syncthreads();
    #pragma unroll 32
    for(int i = 0 ; i < BlockSize ; i++){
        store[y * BlockSize + x] = min(herizonal[y * BlockSize + i] + vertical[i * BlockSize + x] , store[y * BlockSize + x]);
        store[(y + Half) * BlockSize + x] = min(herizonal[(y + Half) * BlockSize + i] + vertical[i * BlockSize + x] , store[(y + Half)* BlockSize + x]);
        store[y * BlockSize + (x + Half)] = min(herizonal[y * BlockSize + i] + vertical[i * BlockSize + (x + Half)] , store[y * BlockSize + (x + Half)]);
        store[(y + Half) * BlockSize + (x + Half)] = min(herizonal[(y + Half) * BlockSize + i] + vertical[i * BlockSize + (x + Half)] , store[(y + Half)* BlockSize + (x + Half)]);
    }
    
    Dist_d[y_real * new_n + x_real] = store[y * BlockSize + x];
    Dist_d[(y_real + Half) * new_n + x_real] = store[(y + Half)* BlockSize + x];
    Dist_d[y_real * new_n + x_real + Half] = store[y * BlockSize + ( x + Half)];
    Dist_d[(y_real + Half) * new_n + ( x_real + Half )] = store[(y + Half) * BlockSize + (x + Half) ];
    
}


void input(char* infile) {

    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    new_n = ceil(n , BlockSize) * BlockSize; // for the purpose to make coalsememory
    Dist = (int*) malloc(new_n * new_n *sizeof(int));

    for (int i = 0; i < new_n; ++i) {
        for (int j = 0; j < new_n ; ++j) {
            if (i == j) {
                Dist[i * new_n + j] = 0;
            } else {
                Dist[i * new_n + j] = INF;
            }
        }
    }


    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0] * new_n + pair[1]] = pair[2];
    }

    /*for(int i = 0 ; i < n ; i++){
        for(int j = 0 ; j < n ; j++){
            printf("%d " , Dist[i * new_n + j]);
        }
        printf("\n");
    }*/
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; ++i) {
		fwrite(Dist + i * new_n , sizeof(int), n , outfile);
	}
    /*for(int i = 0 ; i < n ; i++){
        for(int j = 0 ; j < n ; j++){
            printf("%d " , Dist[i * new_n + j]);
        }
        printf("\n");
    }*/
    fclose(outfile);
}



int main(int argc, char* argv[]) {
    input(argv[1]);
    // int BlockSize = 512;
    int* deviceDist;
    printf("%d \n", new_n);
    size_t size_ = sizeof(int) *new_n * new_n;
    hipHostRegister(Dist,size_, hipHostRegisterDefault);
    hipMalloc(&deviceDist, size_);
	hipMemcpy(deviceDist, Dist, size_, hipMemcpyHostToDevice);

    int total_block_num = new_n / BlockSize;
    dim3 block_num1(1, 1);
    dim3 block_num2(1, total_block_num);
    dim3 block_num3(total_block_num, total_block_num);
    // dim3 thread_each_block(BlockSize,BlockSize);
    // dim3 thread_each_block(BlockSize, 16 );
    dim3 thread_32(32,32);
    // printf("the total block : %d \n" , total_block_num);
    // printf("the n is %d and the new_n is %d \n" ,  n , new_n);
    
    for(int i = 0 ; i < total_block_num ; i++){
        phase1<<<block_num1 , thread_32>>>(deviceDist , i , new_n); // phase 1 
        phase2<<<block_num2 , thread_32>>>(deviceDist , i , new_n); // phase 2
        phase3<<<block_num3 , thread_32>>>(deviceDist , i , new_n); // phase 3
    }
    hipMemcpy(Dist , deviceDist, size_, hipMemcpyDeviceToHost); 
    output(argv[2]);
    return 0;
}