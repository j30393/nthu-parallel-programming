#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <zlib.h>
#include <png.h>

#define Z 2
#define Y 5
#define X 5
#define xBound X / 2
#define yBound Y / 2
#define SCALE 8

int read_png(const char* filename, unsigned char** image, unsigned* height, 
             unsigned* width, unsigned* channels) {

    unsigned char sig[8];
    FILE* infile;
    infile = fopen(filename, "rb");

    fread(sig, 1, 8, infile);
    if (!png_check_sig(sig, 8))
        return 1;   /* bad signature */

    png_structp png_ptr;
    png_infop info_ptr;

    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr)
        return 4;   /* out of memory */
  
    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        return 4;   /* out of memory */
    }

    png_init_io(png_ptr, infile);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, info_ptr);
    int bit_depth, color_type;
    png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

    png_uint_32  i, rowbytes;
    png_bytep  row_pointers[*height];
    png_read_update_info(png_ptr, info_ptr);
    rowbytes = png_get_rowbytes(png_ptr, info_ptr);
    *channels = (int) png_get_channels(png_ptr, info_ptr);

    if ((*image = (unsigned char *) malloc(rowbytes * *height)) == NULL) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        return 3;
    }

    for (i = 0;  i < *height;  ++i)
        row_pointers[i] = *image + i * rowbytes;
    png_read_image(png_ptr, row_pointers);
    png_read_end(png_ptr, NULL);
    return 0;
}

void write_png(const char* filename, png_bytep image, const unsigned height, const unsigned width, 
               const unsigned channels) {
    FILE* fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8,
                 PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
                 PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
    png_write_info(png_ptr, info_ptr);
    png_set_compression_level(png_ptr, 1);

    png_bytep row_ptr[height];
    for (int i = 0; i < height; ++ i) {
        row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
    }
    png_write_image(png_ptr, row_ptr);
    png_write_end(png_ptr, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}

__constant__ char mask[Z][Y][X] = { { { -1, -4, -6, -4, -1 },
                                        { -2, -8, -12, -8, -2 },
                                        { 0, 0, 0, 0, 0 },
                                        { 2, 8, 12, 8, 2 },
                                        { 1, 4, 6, 4, 1 } },
                                      { { -1, -2, 0, 2, 1 },
                                        { -4, -8, 0, 8, 4 },
                                        { -6, -12, 0, 12, 6 },
                                        { -4, -8, 0, 8, 4 },
                                        { -1, -2, 0, 2, 1 } } };

inline __device__ int bound_check(int val, int lower, int upper) {
    if (val >= lower && val < upper)
        return 1;
    else
        return 0;
}

__global__ void sobel(unsigned char *s, unsigned char *t, unsigned height, unsigned width, unsigned channels) {
    /* add shared memo */
    __shared__ unsigned char sR[5][260];
    __shared__ unsigned char sG[5][260];
    __shared__ unsigned char sB[5][260];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y ;
    if (tid >= width) return;
    // printf("%d %d \n" , tid , height );
    // printf("blockidx.x %d blockidx.y %d \n" , blockIdx.x, blockIdx.y );
    // int y = tid;
    // printf("y is %d " , y);
    //  printf("width is %d \n" , width);
    #pragma unroll 5
    for (int v = -yBound; v <= yBound; ++v){
        if(bound_check(y + v, 0, height)){
            sR[v + yBound][threadIdx.x + xBound] = s[channels * (width * (y + v) + tid) + 2];
            sG[v + yBound][threadIdx.x + xBound] = s[channels * (width * (y + v) + tid) + 1];
            sB[v + yBound][threadIdx.x + xBound] = s[channels * (width * (y + v) + tid) + 0];
            if(threadIdx.x == 0){
                if(blockIdx.x != 0){
                    sR[v + yBound][0] = s[channels * (width * (y + v) + tid - 2 ) + 2];
                    sG[v + yBound][0] = s[channels * (width * (y + v) + tid - 2 ) + 1];
                    sB[v + yBound][0] = s[channels * (width * (y + v) + tid - 2 ) + 0];
                    sR[v + yBound][1] = s[channels * (width * (y + v) + tid - 1 ) + 2];
                    sG[v + yBound][1] = s[channels * (width * (y + v) + tid - 1 ) + 1];
                    sB[v + yBound][1] = s[channels * (width * (y + v) + tid - 1 ) + 0];
                }
            }
            else if(threadIdx.x == blockDim.x - 1){
                if(tid + blockDim.x < width){
                    sR[v + yBound][threadIdx.x + xBound + 1] = s[channels * (width * (y + v) + tid + 1) + 2];
                    sG[v + yBound][threadIdx.x + xBound + 1] = s[channels * (width * (y + v) + tid + 1) + 1];
                    sB[v + yBound][threadIdx.x + xBound + 1] = s[channels * (width * (y + v) + tid + 1) + 0];
                    sR[v + yBound][threadIdx.x + xBound + 2] = s[channels * (width * (y + v) + tid + 2) + 2];
                    sG[v + yBound][threadIdx.x + xBound + 2] = s[channels * (width * (y + v) + tid + 2) + 1];
                    sB[v + yBound][threadIdx.x + xBound + 2] = s[channels * (width * (y + v) + tid + 2) + 0];
                }
            }
        }
    }
    __syncthreads();
    //  printf("%d the thread is \n" , threadIdx.x);
    /*   int tid = blockIdx.x * blockDim.x + threadIdx.x;    
    printf("%d %d \n" , tid , height );
    if (tid >= height) return; */


    float val[Z][3];
    #pragma unroll 2
    for (int i = 0; i < Z; ++i) {

        val[i][2] = 0.;
        val[i][1] = 0.;
        val[i][0] = 0.;

        /* Y and X axis of mask */
        #pragma unroll 5
        for (int v = -yBound; v <= yBound; ++v) {
            #pragma unroll 5
            for (int u = -xBound; u <= xBound; ++u) {
                if (bound_check(tid + u, 0, width) && bound_check(y + v, 0, height)) {
                    const unsigned char R = sR[v+2][threadIdx.x+u+2];
                    const unsigned char G = sG[v+2][threadIdx.x+u+2];
                    const unsigned char B = sB[v+2][threadIdx.x+u+2];
                    val[i][2] += R * mask[i][u + xBound][v + yBound];
                    val[i][1] += G * mask[i][u + xBound][v + yBound];
                    val[i][0] += B * mask[i][u + xBound][v + yBound];
                }
            }
        }
    }
    float totalR = 0.;
    float totalG = 0.;
    float totalB = 0.;
    for (int i = 0; i < Z; ++i) {
        totalR += val[i][2] * val[i][2];
        totalG += val[i][1] * val[i][1];
        totalB += val[i][0] * val[i][0];
    }
    totalR = sqrt(totalR) / SCALE;
    totalG = sqrt(totalG) / SCALE;
    totalB = sqrt(totalB) / SCALE;
    const unsigned char cR = (totalR > 255.) ? 255 : totalR;
    const unsigned char cG = (totalG > 255.) ? 255 : totalG;
    const unsigned char cB = (totalB > 255.) ? 255 : totalB;
    t[channels * (width * y + tid) + 2] = cR;
    t[channels * (width * y + tid) + 1] = cG;
    t[channels * (width * y + tid) + 0] = cB;

}


int main(int argc, char **argv) {
    assert(argc == 3);
    unsigned height, width, channels;
    unsigned char *src = NULL, *dst;
    unsigned char *dsrc, *ddst;

    /* read the image to src, and get height, width, channels */
    if (read_png(argv[1], &src, &height, &width, &channels)) {
        std::cerr << "Error in read png" << std::endl;
        return -1;
    }

    dst = (unsigned char *)malloc(height * width * channels * sizeof(unsigned char));
    hipHostRegister(src, height * width * channels * sizeof(unsigned char), hipHostRegisterDefault);

    // hipMalloc(...) for device src and device dst
    hipMalloc(&dsrc, height * width * channels * sizeof(unsigned char));
    hipMalloc(&ddst, height * width * channels * sizeof(unsigned char));

    // hipMemcpy(...) copy source image to device (mask matrix if necessary)
    hipMemcpy(dsrc, src, height * width * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

    // decide to use how many blocks and threads
    int num_threads = 256;
    int blockNum_x = width / num_threads + 1; // modify to have the coalesced memo 
    int blockNum_y = height;
    dim3 num_blocks(blockNum_x, blockNum_y);
    // launch cuda kernel

    sobel << <num_blocks, num_threads>>> (dsrc, ddst, height, width, channels);

    // hipMemcpy(...) copy result image to host
    hipMemcpy(dst, ddst, height * width * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);

    write_png(argv[2], dst, height, width, channels);
    free(src);
    free(dst);
    hipFree(dsrc);
    hipFree(ddst);
    return 0;
}

